#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "./pfc_cuda_device_info.h"
#include "../CudaLib/pfc_cuda_exception.h"
#include <iostream>

using namespace std::literals;

__global__ void cs_kernel(char * const dp_dst, char * const dp_src ,int const size) {
	auto const i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		dp_dst[i] = dp_src[i];
	}
}

int main()
{
	try {
		int count { 0 };
		hipGetDeviceCount(&count);

		if (count > 0)
		{
			hipSetDevice(0);
			auto const  dev_info = pfc::cuda::get_device_info();
			auto const  dev_prop = pfc::cuda::get_device_props();

			std::cout << "device: " << dev_prop.name << std::endl;
			std::cout << "compute capability: " << dev_info.cc_major << '.' << dev_info.cc_minor << std::endl;

			auto const text = "hello world"s;
			auto const size = std::size(text) + 1;
			auto const threads_in_block = 32;
			auto const blocks_in_grid = (size + threads_in_block - 1) / threads_in_block;
			auto const * const hp_src = text.c_str();
			auto * hp_dst = new char [size] {0};

			char * dp_src = nullptr;
			hipMalloc(&dp_src, size);

			char * dp_dst = nullptr;
			hipMalloc(&dp_dst, size);

			hipMemcpy(dp_src, hp_src, size, hipMemcpyHostToDevice);

			cs_kernel <<<blocks_in_grid , threads_in_block>>>(dp_dst, dp_src, size);

			hipDeviceSynchronize();

			hipGetLastError();

			hipMemcpy(hp_dst, dp_dst, size, hipMemcpyDeviceToHost);

			std::cout << "result: '" << hp_dst << "'" << std::endl;

			hipFree(dp_src);
			hipFree(dp_dst);

			delete[] hp_dst;
			hp_dst = nullptr;
		}
	}
	catch (std::exception const & x) {	
	std::cerr << x.what() << '\n';
	}
	return 0;
}

